#include "hip/hip_runtime.h"
#pragma once

#include <hip/hip_runtime.h>
#include <>
#include <device_atomic_functions.h>
#include <stdio.h>
#include "mex.h"

// CUDA error checking macro
#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/* SOURCE */
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s %s %d\n",
            hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ inline bool isMatch(const double* timeSeries, int i, int j, int m, double r)
{
    for (int k = 0; k < m; ++k)
    {
        if (fabs(timeSeries[i + k] - timeSeries[j + k]) > r)
            return false;
    }
    return true;
}

__global__ void countMatches(const double* timeSeries, int N, int m, double r, unsigned long long* d_B)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N - m + 1) return;

    unsigned long long count = 0;
    for (int j = 0; j < N - m + 1; ++j)
    {
        if (i == j) continue; // Exclude self-matches
        if (isMatch(timeSeries, i, j, m, r))
            count++;
    }

    atomicAdd(d_B, count);
}

__global__ void countMatchesExtended(const double* timeSeries, int N, int m, double r, unsigned long long* d_A)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N - m) return;

    unsigned long long count = 0;
    for (int j = 0; j < N - m; ++j)
    {
        if (i == j) continue; // Exclude self-matches
        if (isMatch(timeSeries, i, j, m + 1, r))
            count++;
    }

    atomicAdd(d_A, count);
}

__host__ void sampEnCUDA(const double* h_timeSeries, int N, int m, double r_factor, double& sampEn) {
    // Compute the mean
    double mean = 0.0;
    for (int i = 0; i < N; ++i) mean += h_timeSeries[i];
    mean /= N;

    // Compute the standard deviation
    double std_dev = 0.0;
    for (int i = 0; i < N; ++i) std_dev += (h_timeSeries[i] - mean) * (h_timeSeries[i] - mean);
    std_dev = sqrt(std_dev / N);

    // Calculate tolerance `r` using the standard deviation
    double r = r_factor * std_dev;

    // Allocate device memory
    double* d_timeSeries;
    unsigned long long* d_A;
    unsigned long long* d_B;
    cudaCheckError(hipMalloc((void**)&d_timeSeries, N * sizeof(double)));
    cudaCheckError(hipMalloc((void**)&d_A, sizeof(unsigned long long)));
    cudaCheckError(hipMalloc((void**)&d_B, sizeof(unsigned long long)));

    // Copy data to device
    cudaCheckError(hipMemcpy(d_timeSeries, h_timeSeries, N * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckError(hipMemset(d_A, 0, sizeof(unsigned long long)));
    cudaCheckError(hipMemset(d_B, 0, sizeof(unsigned long long)));

    // Kernel launch parameters
    int threadsPerBlock = 256;
    int blocksPerGrid_m = (N - m + 1 + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGrid_m1 = (N - m + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernels
    countMatches<<<blocksPerGrid_m, threadsPerBlock>>>(d_timeSeries, N, m, r, d_B);
    cudaCheckError(hipGetLastError());

    countMatchesExtended<<<blocksPerGrid_m1, threadsPerBlock>>>(d_timeSeries, N, m, r, d_A);
    cudaCheckError(hipGetLastError());

    // Copy results back to host
    unsigned long long h_A = 0;
    unsigned long long h_B = 0;
    cudaCheckError(hipMemcpy(&h_A, d_A, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(&h_B, d_B, sizeof(unsigned long long), hipMemcpyDeviceToHost));

    // Compute Sample Entropy
    if (h_B == 0 || h_A == 0) {
        mexErrMsgIdAndTxt("sampleEntropy_cuda:zeroCounts", "Sample entropy undefined: zero matches found in patterns.");
    } else {
        sampEn = -log((double)h_A / h_B);
    }

    // Free device memory
    hipFree(d_timeSeries);
    hipFree(d_A);
    hipFree(d_B);
}