#include "hip/hip_runtime.h"
#pragma once

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "mex.h"

// CUDA error checking macro
#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/* SOURCE */
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s %s %d\n",
            hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ inline bool isMatchXY(const double* xSeries, const double* ySeries, int i, int j, int m, double r)
{
    for (int k = 0; k < m; ++k)
    {
        if (fabs(xSeries[i + k] - ySeries[j + k]) > r)
            return false;
    }
    return true;
}

__global__ void computeCiXY(const double* xSeries, const double* ySeries, int N, int m, double r, double* d_Ci)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N - m + 1) return;

    int count = 0;
    int total = N - m + 1;

    for (int j = 0; j < total; ++j)
    {
        if (isMatchXY(xSeries, ySeries, i, j, m, r))
            count++;
    }

    d_Ci[i] = (double)count / total;
}

__host__ void crossApproxEnCUDA(const double* h_xSeries, const double* h_ySeries, int N, int m, double r_factor, double& crossApEn) {
    // Compute the mean and std_dev of both series concatenated
    double mean = 0.0;
    for (int i = 0; i < N; ++i) {
        mean += h_xSeries[i];
        mean += h_ySeries[i];
    }
    mean /= (2 * N);

    // Compute the standard deviation
    double std_dev = 0.0;
    for (int i = 0; i < N; ++i) {
        std_dev += (h_xSeries[i] - mean) * (h_xSeries[i] - mean);
        std_dev += (h_ySeries[i] - mean) * (h_ySeries[i] - mean);
    }
    std_dev = sqrt(std_dev / (2 * N));

    // Calculate tolerance `r` using the standard deviation
    double r = r_factor * std_dev;

    // Allocate device memory
    double* d_xSeries;
    double* d_ySeries;
    double* d_Ci_m;
    double* d_Ci_m1;
    cudaCheckError(hipMalloc((void**)&d_xSeries, N * sizeof(double)));
    cudaCheckError(hipMalloc((void**)&d_ySeries, N * sizeof(double)));
    cudaCheckError(hipMalloc((void**)&d_Ci_m, (N - m + 1) * sizeof(double)));
    cudaCheckError(hipMalloc((void**)&d_Ci_m1, (N - m) * sizeof(double)));

    // Copy data to device
    cudaCheckError(hipMemcpy(d_xSeries, h_xSeries, N * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_ySeries, h_ySeries, N * sizeof(double), hipMemcpyHostToDevice));

    // Kernel launch parameters
    int threadsPerBlock = 256;
    int blocksPerGrid_m = (N - m + 1 + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGrid_m1 = (N - m + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernels
    computeCiXY<<<blocksPerGrid_m, threadsPerBlock>>>(d_xSeries, d_ySeries, N, m, r, d_Ci_m);
    cudaCheckError(hipGetLastError());

    computeCiXY<<<blocksPerGrid_m1, threadsPerBlock>>>(d_xSeries, d_ySeries, N, m + 1, r, d_Ci_m1);
    cudaCheckError(hipGetLastError());

    // Copy results back to host
    double* h_Ci_m = new double[N - m + 1];
    double* h_Ci_m1 = new double[N - m];
    cudaCheckError(hipMemcpy(h_Ci_m, d_Ci_m, (N - m + 1) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(h_Ci_m1, d_Ci_m1, (N - m) * sizeof(double), hipMemcpyDeviceToHost));

    // Compute Φ^xy(m, r) and Φ^xy(m + 1, r)
    double phi_m = 0.0;
    double phi_m1 = 0.0;

    for (int i = 0; i < N - m + 1; ++i)
    {
        if (h_Ci_m[i] > 0)
            phi_m += log(h_Ci_m[i]);
        else
            phi_m += log(1e-10); // Avoid log(0)
    }
    phi_m /= (N - m + 1);

    for (int i = 0; i < N - m; ++i)
    {
        if (h_Ci_m1[i] > 0)
            phi_m1 += log(h_Ci_m1[i]);
        else
            phi_m1 += log(1e-10); // Avoid log(0)
    }
    phi_m1 /= (N - m);

    // Compute cross-approximate entropy
    crossApEn = phi_m - phi_m1;

    // Free memory
    delete[] h_Ci_m;
    delete[] h_Ci_m1;
    hipFree(d_xSeries);
    hipFree(d_ySeries);
    hipFree(d_Ci_m);
    hipFree(d_Ci_m1);
}
